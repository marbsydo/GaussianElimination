#include "hip/hip_runtime.h"
#include "elimination_kernel.h"

void elimination_kernel(float *a, float *b, int n, int kernel) {

	// Copy data to GPU
	int size_a = n * n;
	int size_b = n;
	float *g_a;
	float *g_b;
	hipMalloc((void**)&g_a, size_a * sizeof(float));
	hipMalloc((void**)&g_b, size_b * sizeof(float));
	hipMemcpy(g_a, a, size_a * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size_b * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(1,1,1);
	dim3 dimGrid(1,1,1);

	// Execute kernel on GPU
	switch (kernel) {
	case 0:
		elimination0<<<dimGrid, dimBlock>>>(g_a, g_b, n);
		break;
	case 1:
		dimBlock.x = n;
		dimBlock.y = n;
		//elimination1<<<dimGrid, dimBlock>>>(g_a, g_b, n);
		break;
	}

	// Copy data from GPU
	hipMemcpy(a, g_a, size_a * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b, g_b, size_b * sizeof(float), hipMemcpyDeviceToHost);

	// Tidy up
	hipFree(g_a);
	hipFree(g_b);
	hipDeviceReset();
}

__global__ void elimination0(float *a, float *b, int n) {

}
