#include "hip/hip_runtime.h"
#include "elimination_kernel.h"

#define BLOCK_SIZE 16

float elimination_kernel(float *a, float *b, int size, int kernel) {
	// Start timers
	check("Creating timers");
	hipEvent_t timer1, timer2;
	hipEventCreate(&timer1);
	hipEventCreate(&timer2);
	hipEventRecord(timer1, 0);

	// Copy data to GPU
	int sizeTotal = (size + 1) * size;
	float *g_a, *g_b;
	check("Allocating memory");
	hipMalloc((void**)&g_a, sizeTotal * sizeof(float));
	hipMalloc((void**)&g_b, sizeTotal * sizeof(float));

	check("Copying memory from host to device");
	if (kernel < 8) {
		// Copy a to g_a which the device will use for reference
		hipMemcpy(g_a, a, sizeTotal * sizeof(float), hipMemcpyHostToDevice);
	} else {
		// Copy a to g_b which the device will modify in place
		hipMemcpy(g_b, a, sizeTotal * sizeof(float), hipMemcpyHostToDevice);
	}

	dim3 dimBlock(1,1,1);
	dim3 dimGrid(1,1,1);

	// Memory used for debugging
	float *c = (float*) malloc(sizeTotal * sizeof(float));

	// Execute kernel on GPU
	check("Executing kernel on GPU");
	switch (kernel) {
	case 0:
		elimination0<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 1:
		dimBlock.x = size + 1;
		elimination1<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 2:
		dimBlock.x = size + 1;
		dimBlock.y = size;
		elimination2<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 3:
		dimBlock.x = size + 1;
		dimBlock.y = size;
		elimination3<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 4:
		dimBlock.x = size + 1;
		dimBlock.y = size;
		elimination4<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 5:
		dimBlock.x = BLOCK_SIZE;
		dimBlock.y = BLOCK_SIZE;
		dimGrid.x = 1;
		dimGrid.y = 1;
		//dimGrid.x = (size + 1 - 1) / BLOCK_SIZE + 1;
		//dimGrid.y = (size - 1) / BLOCK_SIZE + 1;
		elimination5<<<dimGrid, dimBlock>>>(g_a, g_b, size);
		break;
	case 6:
		dimBlock.x = size + 1;
		dimBlock.y = size;
		elimination6<<<dimGrid, dimBlock>>>(g_a, g_b, size, 0);
		for (unsigned int i = 1; i < size; i++) {
			hipMemcpy(c, g_b, sizeTotal * sizeof(float), hipMemcpyDeviceToHost);
			printf("Debug %d:\n", i);
			elimination_gold_print_matrix(c, size);
			elimination6<<<dimGrid, dimBlock>>>(g_b, g_b, size, i);
		}
		break;
	case 7:
		printf("Performing 7\n");
		// Each block represents one row
		// Blocks are tiled vertically
		dimBlock.x = size + 1; // Max of 512 threads per block
		dimGrid.x = size;

		elimination7<<<dimGrid, dimBlock>>>(g_a, g_b, size, 0);
		for (unsigned int i = 1; i < size; i++) {
			hipMemcpy(c, g_b, sizeTotal * sizeof(float), hipMemcpyDeviceToHost);
			printf("Debug %d:\n", i);
			elimination_gold_print_matrix(c, size);
			elimination7<<<dimGrid, dimBlock>>>(g_b, g_b, size, i);
		}
		break;
	case 8:
		dimBlock.x = size + 1;
		dimBlock.y = size;

		for (unsigned int i = 0; i < size; i++)
			elimination8_1<<<dimGrid, dimBlock>>>(g_b, size, i);
		elimination8_2<<<dimGrid, dimBlock>>>(g_b, size);
		break;
	case 9:
		dimBlock.x = size + 1;
		dimBlock.y = size;

		elimination9<<<dimGrid, dimBlock>>>(g_b, size);
		break;
	case 10:
		dimBlock.x = size + 1;
		dimBlock.y = size;

		elimination10<<<dimGrid, dimBlock>>>(g_b, size);
		break;
	case 11:
		dimBlock.x = BLOCK_SIZE;
		dimBlock.y = BLOCK_SIZE;
		dimGrid.x = (size + 1 - 1) / BLOCK_SIZE + 1;
		dimGrid.y = (size - 1) / BLOCK_SIZE + 1;

		for (int pivot = 0; pivot < size; pivot++) {
			elimination11_1<<<dimGrid, dimBlock>>>(g_b, size, pivot);
			hipDeviceSynchronize();
		}
		elimination11_2<<<dimGrid, dimBlock>>>(g_b, size);
		break;
	}

	// Copy data from GPU
	check("Copying data from device to host");
	hipMemcpy(b, g_b, sizeTotal * sizeof(float), hipMemcpyDeviceToHost);

	// Tidy up
	check("Freeing memory");
	hipFree(g_a);
	hipFree(g_b);

	// Stop timers
	hipEventRecord(timer2, 0);
	hipEventSynchronize(timer1);
	hipEventSynchronize(timer2);
	float elapsed;
	hipEventElapsedTime(&elapsed, timer1, timer2);

	hipDeviceReset();
	return elapsed;
}

// Very, very naive implementation, identical to CPU code
__global__ void elimination0(float *a, float *b, int size) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;
	float c;

	// The matrix will be modified in place, so first make a copy of matrix a
	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1
		for (xx = 0; xx < size + 1; xx++)
			element(xx, yy) /= pivot;

		// Make all other values in the pivot column be zero
		for (rr = 0; rr < size; rr++) {
			if (rr != yy) {
				c = element(yy, rr);
				for (xx = 0; xx < size + 1; xx++)
					element(xx, rr) -= c * element(xx, yy);
			}
		}
	}
#undef element
}

// Inner xx loops are now parallel
// Uses one block, so limited to 512 threads
// Still uses only global memory
__global__ void elimination1(float *a, float *b, int size) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;

	// The matrix will be modified in place, so first make a copy of matrix a
	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

	xx = threadIdx.x;

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1
		element(xx, yy) /= pivot;

		// Make all other values in the pivot column be zero
		for (rr = 0; rr < size; rr++) {
			if (rr != yy)
				element(xx, rr) -= element(yy, rr) * element(xx, yy);
		}
	}
#undef element
}

// Both xx and rr loops are now in parallel
__global__ void elimination2(float *a, float *b, int size) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;

	// The matrix will be modified in place, so first make a copy of matrix a
	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

	__syncthreads();

	xx = threadIdx.x;
	rr = threadIdx.y;

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1
		element(xx, yy) /= pivot;

		// Make all other values in the pivot column be zero
		if (rr != yy)
			element(xx, rr) -= element(yy, rr) * element(xx, yy);

		__syncthreads();
	}
#undef element
}

// Data is copied in parallel
__global__ void elimination3(float *a, float *b, int size) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;

	xx = threadIdx.x;
	rr = threadIdx.y;

	int tid = rr * (size + 1) + xx;

	// The matrix will be modified in place, so first make a copy of matrix a
	b[tid] = a[tid];

	__syncthreads();

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1
		element(xx, yy) /= pivot;

		// Make all other values in the pivot column be zero
		if (rr != yy)
			element(xx, rr) -= element(yy, rr) * element(xx, yy);

		__syncthreads();
	}
#undef element
}

// Shared memory is used
// However, still limited to matrices of size 22
__global__ void elimination4(float *a, float *b, int size) {
#define element(_x, _y) (*(sdata + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;

	// With a limit of 512 threads per block, and only one block, this results in a maximum
	// of a matrix size 22, which requires (22 + 1) x 22 values
	__shared__ float sdata[(22 + 1) * 22];

	xx = threadIdx.x;
	rr = threadIdx.y;

	int tid = rr * (size + 1) + xx;

	// The matrix will be modified in place, so first make a copy of matrix a
	sdata[tid] = a[tid];

	for (yy = 0; yy < size; yy++) {

		__syncthreads();

		// Make the pivot be 1
		element(xx, yy) /= element(yy, yy);

		__syncthreads();

		// Make all other values in the pivot column be zero
		if (rr != yy)
			element(xx, rr) -= element(yy, rr) * element(xx, yy);
	}

	b[tid] = sdata[tid];
#undef element
}

// Tries to use tiled implementation; does not work
__global__ void elimination5(float *a, float *b, int size) {
#define element(_x, _y) (*(sdata + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;

	__shared__ float sdata[BLOCK_SIZE * BLOCK_SIZE];

	xx = threadIdx.x;
	rr = threadIdx.y;

	int tid = rr * (size + 1) + xx;

	sdata[tid] = a[blockDim.y * BLOCK_SIZE + blockDim.x + tid];

	for (yy = 0; yy < size; yy++) {

		__syncthreads();

		// Make the pivot be 1
		element(xx, yy) /= element(yy, yy);

		__syncthreads();

		// Make all other values in the pivot column be zero
		if (rr != yy)
			element(xx, rr) -= element(yy, rr) * element(xx, yy);
	}

	b[blockDim.y * BLOCK_SIZE + blockDim.x + tid] = sdata[tid];
#undef element
}

// Kernel is invoked once per pivot
// One block, with thread dimensions equal to matrix dimensions
__global__ void elimination6(float *a, float *b, int size, int pivot) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	int x = threadIdx.x;
	int y = threadIdx.y;

	int tid = y * (size + 1) + x;
	b[tid] = a[tid];

	if (y == pivot)
		element(x, y) /= element(pivot, pivot);

	__syncthreads();

	if (y != pivot)
		element(x, y) -= element(pivot, y) * element(x, pivot);

#undef element
}

// Kernel is invoked once per pivot
// Multiple blocks, with dimensions fixed
__global__ void elimination7(float *a, float *b, int size, int pivot) {
#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	element(threadIdx.x, blockDim.x) = 7;
	/*
	int x = threadIdx.x;
	int y = blockDim.x;

	if (x < size + 1 && y < size) {
		int tid = y * (size + 1) + x;

		b[tid] = a[tid];

		if (y == pivot)
			element(x, y) /= element(pivot, pivot);

		__syncthreads();

		if (y != pivot)
			element(x, y) -= element(pivot, y) * element(x, pivot);
	}
	*/
#undef element
}

__global__ void elimination8_1(float *a, int size, int pivot) {
#define element(_x, _y) (*(a + ((_y) * (size + 1) + (_x))))
	int x = threadIdx.x;
	int y = threadIdx.y;

	float cp = element(pivot, y) / element(pivot, pivot);

	if (y != pivot)
		element(x, y) -= cp * element(x, pivot);

#undef element
}

__global__ void elimination8_2(float *a, int size) {
#define element(_x, _y) (*(a + ((_y) * (size + 1) + (_x))))
	int yy = threadIdx.y * (size + 1) + threadIdx.x;
	element(size, yy) /= element(yy, yy);
#undef element
}

// A combination of both 8_1 and 8_2
// This opens up the possibility of using shared memory
__global__ void elimination9(float *a, int size) {
#define element(_x, _y) (*(a + ((_y) * (size + 1) + (_x))))
	int x = threadIdx.x;
	int y = threadIdx.y;

	float cp;

	for (int pivot = 0; pivot < size; pivot++) {

		cp = element(pivot, y) / element(pivot, pivot);

		if (y != pivot)
			element(x, y) -= cp * element(x, pivot);

		__syncthreads();
	}

	int yy = threadIdx.y * (size + 1) + threadIdx.x;
	element(size, yy) /= element(yy, yy);
#undef element
}

// Uses shared memory, but uses only one block
// Limited by amount of shared memory per block
__global__ void elimination10(float *a, int size) {
#define element(_x, _y) (*(sdata + ((_y) * (size + 1) + (_x))))

	int x = threadIdx.x;
	int y = threadIdx.y;
	int tid = y * (size + 1) + x;

	__shared__ float sdata[(22 + 1) * 22]; // Max size that will fit is 22
	sdata[tid] = a[tid];

	float cp;

	for (int pivot = 0; pivot < size; pivot++) {

		cp = element(pivot, y) / element(pivot, pivot);

		if (y != pivot)
			element(x, y) -= cp * element(x, pivot);

		__syncthreads();
	}

	element(size, tid) /= element(tid, tid);

	__syncthreads();

	a[tid] = sdata[tid];
#undef element
}

__global__ void elimination11_1(float *a, int size, int pivot) {
#define element(_x, _y) (*(a + ((_y) * (size + 1) + (_x))))

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > size || y > size - 1)
		return;

	float cp = element(pivot, y) / element(pivot, pivot);

	if (y != pivot)
		element(x, y) -= cp * element(x, pivot);

#undef element
}

__global__ void elimination11_2(float *a, int size) {
#define element(_x, _y) (*(a + ((_y) * (size + 1) + (_x))))

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > size || y > size - 1)
		return;

	int tid = y * (size + 1) + x;

	element(size, tid) /= element(tid, tid);

#undef element
}
