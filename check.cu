#include "check.h"

#define SHOW_ALL_CHECKS true

void check(char* msg) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		fprintf(stderr, "Error: (%s) %s\n", msg, hipGetErrorString(err));
	else if (SHOW_ALL_CHECKS)
		fprintf(stdout, "Log: (%s) %s\n", msg, hipGetErrorString(err));
}
