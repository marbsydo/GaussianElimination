#include "check.h"

#define SHOW_ALL_CHECKS false

void check(char* msg) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		fprintf(stderr, "Error: (%s) %s\n", msg, hipGetErrorString(err));
	else if (SHOW_ALL_CHECKS)
		fprintf(stdout, "Log: (%s) %s\n", msg, hipGetErrorString(err));
}
