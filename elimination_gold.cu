#include "elimination_gold.h"
#include <stdio.h>

// Performs Gauss-Jordan elimination on the CPU; [A]{x]={b}
// Inputs:
//   a -> [A], the matrix of coefficients of size 'n' by 'n'
//   b -> {b}, the vertical matrix of results
//   n -> width/height of 'a', and height of 'b'
// Outputs:
//   Modifies 'a' into the identity matrix
//   Modifies 'b' into the solution for {x}
float elimination_gold(float *a, float *b, int size) {
	// Start timers
	hipEvent_t timer1, timer2;
	hipEventCreate(&timer1);
	hipEventCreate(&timer2);
	hipEventRecord(timer1, 0);

#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;
	float c;

	// The matrix will be modified in place, so first make a copy of matrix a
	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

#ifdef DEBUG
		printf("Matrix before:\n");
		elimination_gold_print_matrix(b, size);
#endif

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1
		for (xx = 0; xx < size + 1; xx++)
			element(xx, yy) /= pivot;

#ifdef DEBUG
		printf("Matrix (Stage 1; Column %d):\n", yy);
		elimination_gold_print_matrix(b, size);
#endif

		// Make all other values in the pivot column be zero
		for (rr = 0; rr < size; rr++) {
			if (rr != yy) {
				c = element(yy, rr);
				for (xx = 0; xx < size + 1; xx++)
					element(xx, rr) -= c * element(xx, yy);
			}
		}

#ifdef DEBUG
		printf("Matrix (Stage 2; Column %d):\n", yy);
		elimination_gold_print_matrix(b, size);
#endif

	}
#undef element

	// Stop timers
	hipEventRecord(timer2, 0);
	hipEventSynchronize(timer1);
	hipEventSynchronize(timer2);
	float elapsed;
	hipEventElapsedTime(&elapsed, timer1, timer2);
	return elapsed;
}

float elimination_gold2(float *a, float *b, int size) {
	// Start timers
	hipEvent_t timer1, timer2;
	hipEventCreate(&timer1);
	hipEventCreate(&timer2);
	hipEventRecord(timer1, 0);

#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;
	float c;

	// The matrix will be modified in place, so first make a copy of matrix a
	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

#ifdef DEBUG
		printf("Matrix before:\n");
		elimination_gold_print_matrix(b, size);
#endif

	for (yy = 0; yy < size; yy++) {
		float pivot = element(yy, yy);

		// Make the pivot be 1

		// We know that pivot / pivot will equal 1, so just set it to 1
		// This line can be commented out, and the final column will still be correct
		element(yy, yy) = 1;

		// Start from yy + 1 instead of 0. The + 1 is because we have calculated done the pivot
		for (xx = yy + 1; xx < size + 1; xx++)
			element(xx, yy) /= pivot;

#ifdef DEBUG
		printf("Matrix (Stage 1; Column %d):\n", yy);
		elimination_gold_print_matrix(b, size);
#endif

		// Make all other values in the pivot column be zero
		for (rr = 0; rr < size; rr++) {
			if (rr != yy) {
				c = element(yy, rr);

				// We know that this value will be zero
				// This line can be commented out, and the final column will still be correct
				element(yy, rr) = 0;

				// Start from yy + 1 instead of 0. The + 1 is because we have already set one value to zero
				for (xx = yy + 1; xx < size + 1; xx++)
					element(xx, rr) -= c * element(xx, yy);
			}
		}

#ifdef DEBUG
		printf("Matrix (Stage 2; Column %d):\n", yy);
		elimination_gold_print_matrix(b, size);
#endif

	}
#undef element

	// Stop timers
	hipEventRecord(timer2, 0);
	hipEventSynchronize(timer1);
	hipEventSynchronize(timer2);
	float elapsed;
	hipEventElapsedTime(&elapsed, timer1, timer2);
	return elapsed;
}

// This method suffers some loss in precision and is also slower
// However, the main loop is simpler
float elimination_gold3(float *a, float *b, int size) {
	// Start timers
	hipEvent_t timer1, timer2;
	hipEventCreate(&timer1);
	hipEventCreate(&timer2);
	hipEventRecord(timer1, 0);

#define element(_x, _y) (*(b + ((_y) * (size + 1) + (_x))))
	unsigned int xx, yy, rr;
	float pivot, c;

	for (unsigned int i = 0; i < (size + 1) * size; i++)
		b[i] = a[i];

#ifdef DEBUG
		printf("Matrix before:\n");
		elimination_gold_print_matrix(b, size);
#endif

	for (yy = 0; yy < size; yy++) {
		pivot = element(yy, yy);

		for (rr = 0; rr < size; rr++) {
			if (rr != yy) {
				c = element(yy, rr);

				// Combine the subtracting and dividing into one operation
				for (xx = yy + 1; xx < size + 1; xx++)
					element(xx, rr) -= c * element(xx, yy) / pivot;
			}

#ifdef DEBUG
		printf("Matrix (Column %d):\n", yy);
		elimination_gold_print_matrix(b, size);
#endif

		}
	}

	// However, one final division is still required for the last column
	for (yy = 0; yy < size; yy++) {
		element(size, yy) /= element(yy, yy);
	}
#undef element

	// Stop timers
	hipEventRecord(timer2, 0);
	hipEventSynchronize(timer1);
	hipEventSynchronize(timer2);
	float elapsed;
	hipEventElapsedTime(&elapsed, timer1, timer2);
	return elapsed;
}

// Prints a matrix in the format of [A]{b}
// Inputs:
//   a -> [A], the matrix of coefficients of size 'n' by 'n'
//   b -> {b}, the vertical matrix of results
//   n -> width/height of 'a', and height of 'b'
// Outputs:
//   Prints out the matrix as a nicely formatted table
void elimination_gold_print_matrix(float *elements, int size) {
	bool front, end;

	for (unsigned int i = 0; i < (size + 1) * size; i++) {
		front = (i % (size + 1) == 0);
		end = (i % (size + 1) == size );

		if (front) printf("[ ");
		if (end) printf("| ");
		printf("%8.4f ", *(elements + i));
		if (end) printf("]\n");
	}
}
